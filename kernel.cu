#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <vector>
#include <iterator>
#include "png.h"
#include "kernel_header.cuh"

extern "C"
PNG contrastChange(PNG input, int size, unsigned int width,
					unsigned int height, float factor) {

	unsigned char *in = 0;
	unsigned char *out = 0;
	hipError_t cudaStatus;
	hipDeviceProp_t prop;

	PNG outputPng;
	outputPng.Create(input.w, input.h);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cout << "No CUDA devices found!\n";
		exit(1);
	}

	hipGetDeviceProperties(&prop, 0);
	std::cout << "Using device: " << prop.name << "\n";

	hipMalloc((void**)&in, size * sizeof(unsigned char));
	hipMalloc((void**)&out, size * sizeof(unsigned char));

	hipMemcpy(in, &input.data[0], size * sizeof(unsigned char), hipMemcpyHostToDevice);
	input.Free();

	contrast<< <width, height >> >(in, out, factor);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << "\n";

		hipFree(in);
		hipFree(out);

		exit(1);
	} else if(cudaStatus == hipSuccess){
		std::cout << "An image has been processed.\n\n";
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Could not synchronize device!\n";
		hipFree(in);
		hipFree(out);
		exit(1);
	}

	auto tmp = new unsigned char[width * height * 4];

	cudaStatus = hipMemcpy(tmp, out, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(in);
	hipFree(out);

	std::copy(&tmp[0], &tmp[width * height * 4], std::back_inserter(outputPng.data));

	delete[] tmp;

	if (cudaStatus != hipSuccess) {
		std::cout << "Copying from the buffer failed.\n";
		exit(1);
	}

	return outputPng;
}