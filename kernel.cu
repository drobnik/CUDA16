#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <vector>
#include <iterator>
#include "png.h"
#include "kernel_header.cuh"

extern "C"
PNG contrastChange(PNG input, int size, unsigned int width,
					unsigned int height, float factor) {

	unsigned char *in = 0;
	unsigned char *out = 0;
	hipError_t cudaStatus;
	hipDeviceProp_t prop;
	PNG outputPng;
	outputPng.Create(input.w, input.h);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cout << "No CUDA devices found!\n";
		exit(1);
	}

	hipGetDeviceProperties(&prop, 0);
	std::cout << "Using device: " << prop.name << "\n";

	hipMalloc((void**)&in, size * sizeof(unsigned char));
	hipMalloc((void**)&out, size * sizeof(unsigned char));

	hipMemcpy(in, &input.data[0], size * sizeof(unsigned char), hipMemcpyHostToDevice);
	input.Free();

	contrast<< <width, height >> >(in, out, factor);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << "\n";

		hipFree(in);
		hipFree(out);

		exit(1);
	} else if(cudaStatus == hipSuccess){
		std::cout << "An image has been processed.\n\n";
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Could not synchronize device!\n";
		hipFree(in);
		hipFree(out);
		exit(1);
	}

	auto tmp = new unsigned char[width * height * 4];

	cudaStatus = hipMemcpy(tmp, out, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(in);
	hipFree(out);

	std::copy(&tmp[0], &tmp[width * height * 4], std::back_inserter(outputPng.data));

	delete[] tmp;

	if (cudaStatus != hipSuccess) {
		std::cout << "Copying from the buffer failed.\n";
		exit(1);
	}

	return outputPng;
}
/*
int main(int arg, char* args[]) {
	contrFactor = 0.5;
	PNG inPng("Lenna.png");
	PNG outPng;
	outPng.Create(inPng.w, inPng.h);

	//store width and height so we can use them for our output image later
	const unsigned int w = inPng.w;
	const unsigned int h = inPng.h;
	//4 because there are 4 color channels R, G, B, and A
	int size = w * h * 4;

	unsigned char *in = 0;
	unsigned char *out = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cout << "No CUDA devices found!" << std::endl;
		exit(1);
	}

	//prints the device the kernel will be running on
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	std::cout << "Using device: " << prop.name << std::endl;

	// Allocate GPU buffers for the images
	hipMalloc((void**)&in, size * sizeof(unsigned char));
	hipMalloc((void**)&out, size * sizeof(unsigned char));

	// Copy image data from host memory to GPU buffers.
	hipMemcpy(in, &inPng.data[0], size * sizeof(unsigned char), hipMemcpyHostToDevice);

	//free the input image because we do not need it anymore
	inPng.Free();

	// Launch a kernel on the GPU with one thread for each element.
	copy << <w, h >> >(in, out, contrFactor);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
		hipFree(in);
		hipFree(out);
		exit(1);
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Could not synchronize device!" << std::endl;
		hipFree(in);
		hipFree(out);
		exit(1);
	}

	//temporary array to store the result from opencl
	auto tmp = new unsigned char[w * h * 4];
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(tmp, out, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipFree(in);
	hipFree(out);

	//copy the data from the temp array to the png
	std::copy(&tmp[0], &tmp[w * h * 4], std::back_inserter(outPng.data));

	//write the image to file
	outPng.Save("cuda_tutorial_2.png");
	//free the iamge's resources since we are done with it
	outPng.Free();

	//free the temp array
	delete[] tmp;

	if (cudaStatus != hipSuccess) {
		std::cout << "Could not copy buffer memory to host!" << std::endl;
		exit(1);
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		std::cout << "Device reset failed!" << std::endl;
		exit(1);
	}

	return 0;
	}*/